#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <ctime>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "csv.h"
#include <sys/time.h>
#include <fstream>
#include <sstream>

struct Order {
    std::string orderId;
    std::string date;
    std::string state;
};

typedef struct {
    struct timeval startTime;
    struct timeval endTime;
} Timer;

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

__device__ void merge(int *arr, int *temp, int start, int middle, int end) {
    int i = start, j = middle, k = start;

    while (i < middle && j < end) {
        if (arr[i] < arr[j]) {
            temp[k++] = arr[i++];
        } else {
            temp[k++] = arr[j++];
        }
    }
    __syncthreads();

    while (i < middle) temp[k++] = arr[i++];
    __syncthreads();
    while (j < end) temp[k++] = arr[j++];
    __syncthreads();
    for (i = start; i < end; i++) {
        arr[i] = temp[i];
    }
    __syncthreads();
}

__global__ void mergeSortKernel(int *arr, int *temp, int n, int width) {
    int thIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int start = thIdx * width * 2;

    if (start < n) {
        int middle = min(start + width, n);
        int end = min(start + 2 * width, n);
        merge(arr, temp, start, middle, end);

    }
}

__device__ int partition(int *arr, int left, int right) {
    int pivot = arr[right];
    int i = (left - 1);

    for (int j = left; j <= right - 1; j++) {
        if (arr[j] < pivot) {
            i++;
            int temp = arr[i];
            arr[i] = arr[j];
            arr[j] = temp;
        }
    }

    int temp = arr[i + 1];
    arr[i + 1] = arr[right];
    arr[right] = temp;

    return (i + 1);
}

__global__ void quickSortKernel(int *arr, int left, int right) {
    if (left < right) {
        int pi = partition(arr, left, right);

        quickSortKernel<<<1, 1>>>(arr, left, pi - 1);
        quickSortKernel<<<1, 1>>>(arr, pi + 1, right);
    }
}

__global__ void bitonicSortGPU(int *arr, int n, int k, int j) {
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ij = idx ^ j;

    if (idx < n && ij > idx) {
        if ((idx & k) == 0) {
            if (arr[idx] > arr[ij]) {
                int temp = arr[idx];
                arr[idx] = arr[ij];
                arr[ij] = temp;
            }
        } else {
            if (arr[idx] < arr[ij]) {
                int temp = arr[idx];
                arr[idx] = arr[ij];
                arr[ij] = temp;
            }
        }
    }
}

__global__ void radixSortGPU(int *arr, int *output, int n, int exp) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    int count[10] = {0};

    for (int i = 0; i < n; i++) {
        count[(arr[i] / exp) % 10]++;
    }
    __syncthreads();

    for (int i = 1; i < 10; i++) {
        count[i] += count[i - 1];
    }
    __syncthreads();

    for (int i = n - 1; i >= 0; i--) {
        output[count[(arr[i] / exp) % 10] - 1] = arr[i];
        count[(arr[i] / exp) % 10]--;
    }
    __syncthreads();
    for (int i = 0; i < n; i++) {
        arr[i] = output[i];
    }
}

int main() {

    int arraySize;
    std::cout << "Select an input array size from below: " << "\n";
    std::cout << "1024" << "\n";
    std::cout << "16384" << "\n";
    std::cout << "131072" << "\n";
    std::cin >> arraySize;

    int inputSize;

    std::string filePath = std::to_string(arraySize) + ".csv";


    std::ifstream file(filePath);

    if (!file.is_open()) {
        std::cerr << "Error opening file" << std::endl;
        return 1;
    }

    std::string line;
    std::getline(file, line);
    std::istringstream s(line);
    std::string field;
    std::vector<int> tempDates;

    while (getline(s, field, ',')) {
        tempDates.push_back(std::stoi(field));
    }

    file.close();

    int dates[tempDates.size()];
    for (size_t i = 0; i < tempDates.size(); ++i) {
        dates[i] = tempDates[i];
    }

    inputSize = tempDates.size();

    int n = inputSize;

    Timer timer;

    hipError_t error_Status;

    int *d_dates, *out_dates;
    error_Status = hipMalloc((void **)&d_dates, n * sizeof(int));
    if (error_Status != hipSuccess) {
        std::cout << "hipMalloc(d_dates) error" << "\n";
    }
    error_Status = hipMalloc((void **)&out_dates, n * sizeof(int));
    if (error_Status != hipSuccess) {
        std::cout << "hipMalloc(out_dates) error" << "\n";
    }

    error_Status = hipMemcpy(d_dates, dates, n * sizeof(int), hipMemcpyHostToDevice);
    if (error_Status != hipSuccess) {
        std::cout << "hipMemcpy error" << "\n";
    }

    int threadsPerBlock = 32;

    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    // input from the USER (1. Merge, 2. Radix )
    int choice;
    std::cout << "Select an option from below: " << "\n";
    std::cout << "1. Merge Sort" << "\n";
    std::cout << "2. Quick Sort" << "\n";
    std::cout << "3. Bitnoic Sort" << "\n";
    std::cout << "4. Radix Sort" << "\n";

    std::cin >> choice;

    switch(choice) {
        case 1:
        {
            startTime(&timer);

            for (int width = 1; width < n; width *= 2) {
                mergeSortKernel<<<numBlocks, threadsPerBlock>>>(d_dates, out_dates, n, width);
                hipDeviceSynchronize();
            }

            stopTime(&timer);
            
            std::cout << "Merge Sort: <INPUT SIZE: " << n << "> and <THREADS/BLOCK: " << threadsPerBlock << ">" << "\n" << "Elapsed time: " << elapsedTime(timer) << " sec\n";
            break;
        }

        case 2:
        {
            startTime(&timer);
            quickSortKernel<<<1, 1>>>(d_dates, 0, n - 1);
            hipDeviceSynchronize();
            stopTime(&timer);
            std::cout << "Quick Sort: <INPUT SIZE: " << n << "> and <THREADS/BLOCK: " << threadsPerBlock << ">" << "\n" << "Elapsed time: " << elapsedTime(timer) << " sec\n";
            break;
        }
            

        case 3:
        {
            startTime(&timer);
            for (int k = 2; k <= n; k <<= 1) {
                for (int j = k >> 1; j > 0; j >>= 1) {
                    bitonicSortGPU<<<numBlocks, threadsPerBlock>>>(d_dates, n, k, j);
                    hipDeviceSynchronize();
                }
            }
            stopTime(&timer);
            std::cout << "Bitonic Sort: <INPUT SIZE: " << n << "> and <THREADS/BLOCK: " << threadsPerBlock << ">" << "\n" << "Elapsed time: " << elapsedTime(timer) << " sec\n";
            break;
        }
            

        case 4:
        {
            startTime(&timer);
            int maxNum = *std::max_element(dates, dates + n);
            for (int exp = 1; maxNum / exp > 0; exp *= 10) {
                radixSortGPU<<<numBlocks, threadsPerBlock>>>(d_dates, out_dates, n, exp);
                hipDeviceSynchronize();
            }
            stopTime(&timer);
            std::cout << "Radix Sort: <INPUT SIZE: " << n << "> and <THREADS/BLOCK: " << threadsPerBlock << ">" << "\n" << "Elapsed time: " << elapsedTime(timer) << " sec\n";
            break;
        }
    }   

    error_Status = hipMemcpy(dates, d_dates, n * sizeof(int), hipMemcpyDeviceToHost);

    if (error_Status!=hipSuccess)
    {
        std::cout << "hipMemcpy 2" << "\n";
    }


    // Verification

    // std::cout << "Sorted Dates:\n";
    // for (int i = 0; i < n; i++) {
    //     std::cout << dates[i] << "\n";
    // }

    hipFree(out_dates);
    hipFree(d_dates);
    return 0;
}
